#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>
//CUDA RunTime API
#include <hip/hip_runtime.h>

#define MATRIX_SIZE 1024

void printDeviceProp(const hipDeviceProp_t &prop)
{
    printf("Device Name : %s.\n", prop.name);
    printf("totalGlobalMem : %d.\n", prop.totalGlobalMem);
    printf("sharedMemPerBlock : %d.\n", prop.sharedMemPerBlock);
    printf("regsPerBlock : %d.\n", prop.regsPerBlock);
    printf("warpSize : %d.\n", prop.warpSize);
    printf("memPitch : %d.\n", prop.memPitch);
    printf("maxThreadsPerBlock : %d.\n", prop.maxThreadsPerBlock);
    printf("maxThreadsDim[0 - 2] : %d %d %d.\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
    printf("maxGridSize[0 - 2] : %d %d %d.\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
    printf("totalConstMem : %d.\n", prop.totalConstMem);
    printf("major.minor : %d.%d.\n", prop.major, prop.minor);
    printf("clockRate : %d.\n", prop.clockRate);
    printf("textureAlignment : %d.\n", prop.textureAlignment);
    printf("deviceOverlap : %d.\n", prop.deviceOverlap);
    printf("multiProcessorCount : %d.\n", prop.multiProcessorCount);
}

//CUDA Initialization
bool InitCUDA()
{
    int count;
    hipGetDeviceCount(&count);
    if (count == 0) 
    {
        fprintf(stderr, "There is no device.\n");
        return false;
    }

    int i;
    for (i = 0; i < count; i++) 
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, i);
        printDeviceProp(prop);
        if (hipGetDeviceProperties(&prop, i) == hipSuccess) 
        {
            if (prop.major >= 1) 
            {
            break;
            }
        }
    }
    if (i == count) 
    {
        fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
        return false;
    }
    hipSetDevice(i);
    return true;
}

// Generate Random Matrix Elements
void matgen(float* a, int n)
{
    int i, j;
    for (i = 0; i < n; i++)
    {
        for (j = 0; j < n; j++)
        {
            a[i * n + j] = (float)rand() / RAND_MAX + (float)rand() / (RAND_MAX * RAND_MAX);
        }
    }
}

/* Task 1 & 2: Implement Your Kernel Function Here */
__global__ static void matMultCUDA(const float* a, const float* b, float* c, int n)
{
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    if((row < n) && (col < n))
    {
        float localv = 0;
        for(int k = 0; k< n ; k++)//each thread compute a row of a and a col of b
        {
            localv += a[row *n + k] * b[k * n+ col];
        }
        c[row * n +col] = localv;
    }
}

int main()
{
    if (!InitCUDA()) return 0; 
    hipError_t err;
    float *a, *b, *c, *d;

    int n = MATRIX_SIZE;

    a = (float*)malloc(sizeof(float)* n * n); 
    b = (float*)malloc(sizeof(float)* n * n); 
    c = (float*)malloc(sizeof(float)* n * n); 
    d = (float*)malloc(sizeof(float)* n * n);

    srand(0);

    matgen(a, n);
    matgen(b, n);

    float *cuda_a, *cuda_b, *cuda_c;

    /* Task: Memory Allocation */
    hipMalloc(&cuda_a, n * n * sizeof(float));

	hipMalloc(&cuda_b, n * n * sizeof(float));

	hipMalloc(&cuda_c, n * n * sizeof(float));


    /* Task: CUDA Memory Copy from Host to Device */
    hipMemcpy(cuda_a, a, n * n * sizeof(float), hipMemcpyHostToDevice);


	hipMemcpy(cuda_b, b, n * n * sizeof(float), hipMemcpyHostToDevice);


    /* Task: Number of Blocks and Threads && Dimention*/

    // 
    int blockSize = 32; // 
    int numBlocks = (n + blockSize - 1) / blockSize; // 
    dim3 dimGrid(numBlocks,numBlocks,1);
    dim3 dimBlock(blockSize,blockSize,1);

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);

    // Kernel Execution
    matMultCUDA << < dimGrid, dimBlock >> >(cuda_a , cuda_b , cuda_c , n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("GPU Time elapsed: %f ms \n",milliseconds);

    /* Task: CUDA Memory Copy from Device to Host */
    hipMemcpy(c, cuda_c, n * n* sizeof(float), hipMemcpyDeviceToHost);
    
    //Free
    hipFree(cuda_a);
    hipFree(cuda_b);
    hipFree(cuda_c);


 
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    // CPU Implementation of MatMul
    for (int i = 0; i < n; i++)
    {
        for (int j = 0; j < n; j++)
        { 
            double t = 0;
            for (int k = 0; k < n; k++)
            { 
                t += a[i * n + k] * b[k * n + j]; 
            } 
            //printf(" %f",t);
            d[i * n + j] = t; 
            
        } 
    }
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    milliseconds = 0.0f;
    hipEventElapsedTime(&milliseconds, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("CPU Time elapsed: %f ms \n",milliseconds);
    // printf(" CPU result  \n");

    // for (int i = 0; i < n; i++) 
    // {
    //     for (int j = 0; j < n; j++) 
    //     {
    //         float x = c[i * n + j];
    //         printf(" %f",x);
    //     } 
    // }
    // printf(" GPU result  \n");
    // Check the accuracy of GPU results with CPU results
    float max_err = 0;
    float average_err = 0; 
    for (int i = 0; i < n; i++) 
    {
        for (int j = 0; j < n; j++) 
        {
            if (d[i * n + j] != 0)
            { 
                float err = fabs((c[i * n + j] - d[i * n + j]) / d[i * n + j]);
                if (max_err < err) max_err = err; 
                average_err += err; 
            } 
        } 
    }
    printf("Max error: %g Average error: %g\n",max_err, average_err / (n * n));

    hipError_t cudaErr = hipGetLastError();
    if (cudaErr != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(cudaErr));
        return 1; // 
    }

    return 0;
}